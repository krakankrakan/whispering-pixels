
#include <hip/hip_runtime.h>
__global__ void victimKernel(int* a, int* b, int* c, int *d, int *e) {
    d[blockIdx.x * blockDim.x + threadIdx.x] = a[blockIdx.x * blockDim.x + threadIdx.x];
}

void call_victimKernel(size_t threadgroups, size_t threadgroupsize, int* d_a, int* d_b, int* d_c, int* d_d, int* d_e) {
    victimKernel<<<threadgroups, threadgroupsize>>>(d_a, d_b, d_c, d_d, d_e);
}